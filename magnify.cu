#include "hip/hip_runtime.h"
// ͷ�ļ�
#include <helper_gl.h>
#if defined (__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#ifndef glutCloseFunc
#define glutCloseFunc glutWMCloseFunc
#endif
#else
#include <GL/freeglut.h>
#endif

#include <stdio.h> 
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
#include <>
#include <vector>


// �Զ���ͷ�ļ�
#include <draw_graph_2d.h>

using namespace std;

// focal area
vector<int> focalArea;


//const float testCons = 0.842;
extern const float pointSize = 4.0;
extern const float edgeWidth = 1.0;
extern const int   windowSize = 800;

// �Ŵ���
extern float m_factor;

extern int vertexNum;
extern int edgeNum;
// ��ʼ��������
extern float *vertexX, *vertexY;
// ������������
extern float *posX, *posY;
extern float *resultPosX, *resultPosY;
extern float *targetVertexX, *targetVertexY;

// ��������
extern float focalPointX, focalPointY;

// �Ƿ��ڷŴ�״̬
extern BOOLEAN isMagnified;

extern BOOLEAN *edge;

// �����ݶȷ���ʹ�õ����������
extern float *matrixA;
extern float *vectorB_X, *vectorB_Y;
extern float *cons_vectorB_X, *cons_vectorB_Y;

extern int *tmpEdge;


// CUDA����
// �����������
hipError_t addWithCuda(float *c, const float *a, const float *b, unsigned int size);

// �����������
hipError_t minusWithCuda(float *c, const float *a, const float *b, unsigned int size);

// �����г˻�
hipError_t mulWithCuda(const float *a, const float *b, float *result, const int M, const int N, const int S);

// �ͷ����пռ�
void freeAll();

// ��������constraints
void computeAllCons();


// �����ݶȷ���������꣨�����ã�
void conjGradientSolver() {
	//printf("conj!!!!! %6.2f\n", resultPosX[0]);

	// ��ȡ�˴�time constraints
	for (int i = 0; i < vertexNum; i++) {
		vectorB_X[i] = cons_vectorB_X[i] + resultPosX[i] * 2 * REAL_AXIS;
		vectorB_Y[i] = cons_vectorB_Y[i] + resultPosY[i] * 2 * REAL_AXIS;
	}


	// ��ʼ���в������ȱ���
	float *tmpVector = new float[vertexNum];
	float *tmpX = new float[vertexNum];
	float *tmpY = new float[vertexNum];
	float *vectorD = new float[vertexNum];
	float *vectorR = new float[vertexNum];
	
	::memset(tmpX, 0, vertexNum * sizeof(float));
	::memset(tmpY, 0, vertexNum * sizeof(float));
	::memset(vectorD, 0, vertexNum * sizeof(float));

	// ����X�����ϵ�����
	for (int i = 0; i < vertexNum; i++) {
		vectorR[i] = -vectorB_X[i];
	}

	for (int nor = 0; nor < vertexNum * 3; nor++) {
		float denom = 0, num = 0;
		float denom2 = 0, num2 = 0;

		// ����r��ת�ó���r
		/*for (int i = 0; i < vertexNum; i++) {
			denom += vectorR[i] * vectorR[i];
		}*/
		mulWithCuda(vectorR, vectorR, &denom, 1, vertexNum, 1);


		// ����в�r = Ax-b
		/*for (int i = 0; i < vertexNum; i++) {
			float cur = 0;
			for (int j = 0; j < vertexNum; j++) {
				cur += matrixA[i * vertexNum + j] * tmpX[j];
			}
			vectorR[i] = cur - vectorB_X[i];
		}*/
		mulWithCuda(matrixA, tmpX, tmpVector, vertexNum, vertexNum, 1);
		minusWithCuda(vectorR, tmpVector, vectorB_X, vertexNum);


		//����r��ת�ó���r
		/*for (int i = 0; i < vertexNum; i++) {
			num += vectorR[i] * vectorR[i];
		}*/
		mulWithCuda(vectorR, vectorR, &num, 1, vertexNum, 1);

		if (num < 0.000001) {
			//printf("x is break\n");
			break;
		}

		//���㷽������d
		for (int i = 0; i < vertexNum; i++) {
			vectorD[i] = -vectorR[i] + num / denom * vectorD[i];
		}

		//����d��ת�ó���r
		/*for (int i = 0; i < vertexNum; i++) {
			num2 += vectorD[i] * vectorR[i];
		}*/
		mulWithCuda(vectorD, vectorR, &num2, 1, vertexNum, 1);

		//����d��ת�ó���A����d���
		/*for (int i = 0; i < vertexNum; i++) {
			float cur = 0;
			for (int j = 0; j < vertexNum; j++) {
				cur += matrixA[i * vertexNum + j] * vectorD[j];
			}
			denom2 += cur * vectorD[i];
		}*/
		mulWithCuda(matrixA, vectorD, tmpVector, vertexNum, vertexNum, 1);
		mulWithCuda(vectorD, tmpVector, &denom2, 1, vertexNum, 1);

		//���㲽��
		double a = -num2 / denom2;
		for (int i = 0; i < vertexNum; i++) {
			tmpX[i] += a * vectorD[i];
		}

		//printf("all X with CUDA!\n");
	}


	// ����y�����ϵ�����
	::memset(vectorD, 0, vertexNum * sizeof(float));
	for (int i = 0; i < vertexNum; i++) {
		vectorR[i] = -vectorB_Y[i];
	}

	for (int nor = 0; nor < vertexNum * 3; nor++) {
		float denom = 0, num = 0;
		float denom2 = 0, num2 = 0;

		// ����r��ת�ó���r
		mulWithCuda(vectorR, vectorR, &denom, 1, vertexNum, 1);

		// ����в�r = Ax-b

		mulWithCuda(matrixA, tmpY, tmpVector, vertexNum, vertexNum, 1);
		minusWithCuda(vectorR, tmpVector, vectorB_Y, vertexNum);

		//����r��ת�ó���r
		mulWithCuda(vectorR, vectorR, &num, 1, vertexNum, 1);

		if (num < 0.000001) {
			//printf("y is break\n");
			break;
		}

		//���㷽������d
		for (int i = 0; i < vertexNum; i++) {
			vectorD[i] = -vectorR[i] + num / denom * vectorD[i];
		}

		//����d��ת�ó���r
		mulWithCuda(vectorD, vectorR, &num2, 1, vertexNum, 1);

		//����d��ת�ó���A����d���
		mulWithCuda(matrixA, vectorD, tmpVector, vertexNum, vertexNum, 1);
		mulWithCuda(vectorD, tmpVector, &denom2, 1, vertexNum, 1);

		//���㲽��
		double a = -num2 / denom2;
		for (int i = 0; i < vertexNum; i++) {
			tmpY[i] += a * vectorD[i];
		}

		//printf("all Y with CUDA!\n");
	}

	for (int i = 0; i < vertexNum; i++) {
		resultPosX[i] = tmpX[i] / REAL_AXIS;
		resultPosY[i] = tmpY[i] / REAL_AXIS;
		//printf("resultPosX %d is %6.2f\n", i, resultPosX[i]);
	}

	// free

	//delete struConsVecX, struConsVecY;
	delete tmpX, tmpY, tmpVector;
	delete vectorD;
	delete vectorR;
}


// ����targetGraph����������
void calTargetGraph() {

	float disX, disY;
	float focalX = focalPointX * REAL_AXIS;
	float focalY = focalPointY * REAL_AXIS;

	focalArea.clear();

	for (int i = 0; i < vertexNum; i++) {
		disX = posX[i] * REAL_AXIS - focalX;
		disY = posY[i] * REAL_AXIS - focalY;

		// ������ĳ���غ�
		if ((fabs(disX) <= EPSINON) && (fabs(disY) <= EPSINON)) {
			targetVertexX[i] = focalX;
			targetVertexY[i] = focalY;
		}
		else if (fabs(disX) <= EPSINON) {
			targetVertexX[i] = focalX;
			float b_iY = (posY[i] > focalPointY) ? 0.9 * REAL_AXIS : -0.9 * REAL_AXIS;
			float b_i = (posY[i] * REAL_AXIS - focalY) / (b_iY - focalY);
			float B_I = (m_factor + 1) * b_i / (m_factor * b_i + 1);

			targetVertexY[i] = focalY + (b_iY - focalY) * B_I;
		}
		else if (fabs(disY) <= EPSINON) {
			targetVertexY[i] = focalY;
			float b_iX = (posX[i] > focalPointX) ? 0.9 * REAL_AXIS : -0.9 * REAL_AXIS;
			float b_i = (posX[i] * REAL_AXIS - focalX) / (b_iX - focalX);
			float B_I = (m_factor + 1) * b_i / (m_factor * b_i + 1);

			targetVertexX[i] = focalX + (b_iX - focalX) * B_I;
		}
		else {
			float b_iX = (disX > 0) ? 0.9 * REAL_AXIS : -0.9 * REAL_AXIS;
			float b_iY = (b_iX - focalX) * disY / disX;

			if (fabs(b_iY) > 0.9 * REAL_AXIS) {
				b_iY = (disY > 0) ? 0.9 * REAL_AXIS : -0.9 * REAL_AXIS;
				b_iX = focalX + ((b_iY - focalY) / (disY / disX));
			}

			float b_i = (posX[i] * REAL_AXIS - focalX) / (b_iX - focalX);
			float B_I = (m_factor + 1) * b_i / (m_factor * b_i + 1);

			targetVertexX[i] = focalX + (b_iX - focalX) * B_I;
			targetVertexY[i] = focalY + (b_iY - focalY) * B_I;
			/*
						printf("focalX is %f, focalY is %f\n", focalX, focalY);
						printf("disX is %f, disY is %f\n", disX, disY);
						targetVertexX[i] = b_iX;
						targetVertexY[i] = b_iY;
			*/
		}

		// ��¼focal area��ĵ㼯
		float length = sqrt((targetVertexX[i] - focalX) * (targetVertexX[i] - focalX) + 
						    (targetVertexY[i] - focalY) * (targetVertexY[i] - focalY));

		if (length < 0.3 * REAL_AXIS)
			focalArea.push_back(i);

	}

	/*for (int i = 0; i < focalArea.size(); i++) {
		cout <<"focal area: " << focalArea[i] << endl;
	}*/
}

// ������λ�����ݵĺ���
void calculateNewPos() {

	calTargetGraph();

	computeAllCons();

	//conjGradientSolver();


	/*for (int i = 0; i < vertexNum; i++) {
		if (fabs(targetVertexX[i]) < REAL_AXIS) {
			resultPosX[i] = targetVertexX[i] / REAL_AXIS;
		}
		else if (targetVertexX[i] > 0) {
			resultPosX[i] = 0.9;
		}
		else {
			resultPosX[i] = -0.9;
		}
		if (fabs(targetVertexY[i]) < REAL_AXIS) {
			resultPosY[i] = targetVertexY[i] / REAL_AXIS;
		}
		else if (targetVertexY[i] > 0) {
			resultPosY[i] = 0.9;
		}
		else {
			resultPosY[i] = -0.9;
		}
	}*/

}

// ��������constraints
void computeAllCons() {
	//printf("computeAllCons\n");

	::memset(matrixA, 0, vertexNum * vertexNum * sizeof(float));
	::memset(vectorB_X, 0, vertexNum * sizeof(float));
	::memset(vectorB_Y, 0, vertexNum * sizeof(float));

	//float constraints;

	// �����߼�, ˫���ȸ�������߾���
	double struConsX, struConsY, length, e_x, e_y;

	// ������
	float w_Stru = 4.0, w_Focal = 2.0, w_Time = 1.0;

	/************************************************/
	// Structure constraints
	/************************************************/
	for (int i = 0; i < vertexNum; i++) {
		for (int j = i; j < vertexNum; j++) {
			if (edge[i * vertexNum + j] == TRUE) {
				matrixA[i * vertexNum + i] += 2.0 * w_Stru;
				matrixA[j * vertexNum + j] += 2.0 * w_Stru;
				matrixA[i * vertexNum + j] -= 2.0 * w_Stru;
				matrixA[j * vertexNum + i] -= 2.0 * w_Stru;
				if (fabs(posX[i] - posX[j]) < EPSINON) {
					struConsX = 0.0;

					//int factor = (posY[i] - posY[j]) ? 1 : -1;
					struConsY = (targetVertexY[i] - targetVertexY[j]) * w_Stru;
					//printf("struConsY%d,%d is %6.2f, %6.2f\n", i, j, targetVertexY[i] - targetVertexY[j], struConsY);
				}
				else {
					double tmpLength = sqrt((posY[i] - posY[j]) * (posY[i] - posY[j]) + (posX[i] - posX[j]) * (posX[i] - posX[j]));
					e_x = (posX[i] - posX[j]) / tmpLength;
					e_y = (posY[i] - posY[j]) / tmpLength;

					length = sqrt((targetVertexX[i] - targetVertexX[j]) * (targetVertexX[i] - targetVertexX[j])
						+ (targetVertexY[i] - targetVertexY[j]) * (targetVertexY[i] - targetVertexY[j]));

					struConsX = e_x * length * w_Stru;
					struConsY = e_y * length * w_Stru;

				}

				vectorB_X[i] += (float)struConsX * 2;
				vectorB_X[j] -= (float)struConsX * 2;
				vectorB_Y[i] += (float)struConsY * 2;
				vectorB_Y[j] -= (float)struConsY * 2;
			}
		}
	}


	/************************************************/
	// Readability constraints
	/************************************************/
	for (int m = 0; m < focalArea.size(); m++) {
		for (int n = m + 1; n < focalArea.size(); n++) {
			int i = focalArea[m];
			int j = focalArea[n];

			matrixA[i * vertexNum + i] += 2.0 * w_Focal;
			matrixA[j * vertexNum + j] += 2.0 * w_Focal;
			matrixA[i * vertexNum + j] -= 2.0 * w_Focal;
			matrixA[j * vertexNum + i] -= 2.0 * w_Focal;

			float length = sqrt((targetVertexX[i] - targetVertexX[j]) * (targetVertexX[i] - targetVertexX[j]) +
				(targetVertexY[i] - targetVertexY[j]) * (targetVertexY[i] - targetVertexY[j]));

			float length_overlapped = (pointSize * 2 / windowSize + 0.04) * REAL_AXIS;

			// ���㷢���غ�
			if (length < length_overlapped) {
				printf("i %d and j %d is overlapped!\n", i, j);

				if (fabs(posX[i] - posX[j]) < EPSINON) {
					struConsX = 0.0;

					int factor = (posY[i] - posY[j]) ? 1 : -1;
					struConsY = length_overlapped * factor * w_Focal;
				}
				else {
					double tmpLength = sqrt((posY[i] - posY[j]) * (posY[i] - posY[j]) + (posX[i] - posX[j]) * (posX[i] - posX[j]));
					e_x = (posX[i] - posX[j]) / tmpLength;
					e_y = (posY[i] - posY[j]) / tmpLength;

					struConsX = e_x * length_overlapped * w_Focal;
					struConsY = e_y * length_overlapped * w_Focal;

				}
			}
			// ���ص����
			else {
				if (fabs(posX[i] - posX[j]) < EPSINON) {
					struConsX = 0.0;

					int factor = (posY[i] - posY[j]) ? 1 : -1;
					struConsY = length * factor * w_Focal;
				}
				else {
					double tmpLength = sqrt((posY[i] - posY[j]) * (posY[i] - posY[j]) + (posX[i] - posX[j]) * (posX[i] - posX[j]));
					e_x = (posX[i] - posX[j]) / tmpLength;
					e_y = (posY[i] - posY[j]) / tmpLength;

					struConsX = e_x * length * w_Focal;
					struConsY = e_y * length * w_Focal;

				}
			}


			vectorB_X[i] += (float)struConsX * 2;
			vectorB_X[j] -= (float)struConsX * 2;
			vectorB_Y[i] += (float)struConsY * 2;
			vectorB_Y[j] -= (float)struConsY * 2;
		}
	}

	/************************************************/
	// Time Corhenrecy constraints
	/************************************************/
	for (int i = 0; i < vertexNum; i++) {
		matrixA[i * vertexNum + i] += 2.0 * w_Time;
	}

	for (int i = 0; i < vertexNum; i++) {
		cons_vectorB_X[i] = vectorB_X[i];
		cons_vectorB_Y[i] = vectorB_Y[i];
	}
}

// �ͷ�����ȫ�ֱ����ռ�
void freeAll() {
	printf("all has been free!\n");

	delete vertexX, vertexY;

	delete posX, posY;
	delete resultPosX, resultPosY;
	delete targetVertexX, targetVertexY;

	// �ͷŹ����ݶȷ����������������
	delete matrixA;
	delete vectorB_X, vectorB_Y;

	delete edge;
	delete tmpEdge;
}


// ������
int main(int argc, char *argv[]) {
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);

	windowInit();

	// ��ʾͼ��
	glutDisplayFunc(&myDisplay);
	glutMouseFunc(mouse);
	glutMainLoop();

	// �ͷ��ڴ�
	freeAll();

	return 0;
}



/***************************************************************************************
****************************************************************************************
								CUDA����
****************************************************************************************
****************************************************************************************/


// add kernel
__global__ void addKernel(float *c, const float *a, const float *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float *c, const float *a, const float *b, unsigned int size)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}


// minus kernel
__global__ void minusKernel(float *c, const float *a, const float *b)
{
	int i = threadIdx.x;
	c[i] = a[i] - b[i];
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t minusWithCuda(float *c, const float *a, const float *b, unsigned int size)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	minusKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}


/* gpuMatMultKernel��GPU�¾���˷��˺���
*  a:��һ������ָ�룬��ʾa[M][N]
*  b:�ڶ�������ָ�룬��ʾb[N][S]
*  result:������󣬱�ʾresult[M][S]
*/
__global__ void gpuMatMultKernel(const float *a, const float *b, float *result, const int M, const int N, const int S)
{
	//int threadId = threadIdx.x + blockIdx.x * blockDim.x;

	int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < M * S)
	{
		int row = threadId / S;
		int column = threadId % S;

		result[threadId] = 0;
		for (int i = 0; i < N; i++)
		{
			result[threadId] += a[row * N + i] * b[i * S + column];
		}
	}
}


// ����CUDA����GPU����˷��˺���
hipError_t mulWithCuda(const float *a, const float *b, float *result, const int M, const int N, const int S)
{
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_result = 0;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_result, M * S * sizeof(float));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a, M * N * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, N * S * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");
		goto Error;
	}

	/*hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);*/

	/*const int THREADNUM = 256;
	const int BLOCKNUM = (M * S + 255) / 256;*/

	const int BLOCK_SIZE = 16;
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((S + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
	gpuMatMultKernel << <grid, block >> > (dev_a, dev_b, dev_result, M, N, S);
	//gpuMatMultWithSharedKernel<16> << <grid, block >> >(dev_a, dev_b, dev_result, M, N, S);
	//printf("This is NOT shared kernel!\n");

	/*hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe runing time of GPU on Mat Multiply is %f seconds.\n", elapsedTime / 1000.0);*/

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
		goto Error;
	}

Error:
	//printf("is free!\n");
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_result);

	return cudaStatus;
}


